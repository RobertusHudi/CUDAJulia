#include <stdio.h>
#include <hip/hip_runtime.h>

#define N 16384
//#define ll long long 

__global__ void matrix_mul(int *a,int *b, int *c, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int sum = 0;
    if (i < n && j < n) {
        for (int k = 0; k < n; k++)
            sum += a[i * n + k] * b[k * n + j];
        c[i * n + j] = sum;
    }
}

int main() {
    int n = N;
    int *a, *b, *c;
    int *d_a, *d_b, *d_c;
    int size = n * n * sizeof(int);

    a = (int *)malloc(size);
    b = (int *)malloc(size);
    c = (int *)malloc(size);

    for (int i = 0; i < n; i++)
        for (int j = 0; j < n; j++) {
            a[i * n + j] = i + j;
            if(i==j)b[i * n + j] = 1;
        }

    // for (int i = 0; i < n; i++) {
    //     for (int j = 0; j < n; j++)
    //         printf("%d ", b[i * n + j]);
    //     printf("\n\n");
    // }

    // for (int i = 0; i < n; i++) {
    //     for (int j = 0; j < n; j++)
    //         printf("%d ", a[i * n + j]);
    //     printf("\n\n");
    // }

    hipMalloc((void **)&d_a, size);
    hipMalloc((void **)&d_b, size);
    hipMalloc((void **)&d_c, size);

    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

    int N_THREADS = 32;
    int N_BLOCKS = (n + N_THREADS - 1) / N_THREADS;

    dim3 threads(N_THREADS, N_THREADS);
    dim3 blocks(N_BLOCKS, N_BLOCKS);


    // dim3 blockSize(N, N);
    // dim3 gridSize((n + N - 1) / N, (n + N - 1) / N);
    matrix_mul<<<blocks, threads>>>(d_a, d_b, d_c, n);

    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

    // for (int i = 0; i < n; i++) {
    //     for (int j = 0; j < n; j++)
    //         printf("%d ", c[i * n + j]);
    //     printf("\n");
    // }

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    free(a);
    free(b);
    free(c);

    return 0;
}
